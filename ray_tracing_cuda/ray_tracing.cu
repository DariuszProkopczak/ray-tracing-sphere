
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <omp.h>
#define THREADS_PER_BLOCK 512

typedef struct vector {
	double x;
	double y;
	double z;
} Vector;

// A 63−bit LCG
// Returns a double precision value from a uniform distribution
// between 0.0 and 1.0 using a caller −owned state variable . 
__device__ double LCG_random_double(uint64_t * seed)
{
	const uint64_t m = 9223372036854775808ULL; // 2ˆ63
	const uint64_t a = 2806196910506780709ULL; 
	const uint64_t c = 1ULL;
	 
	*seed = (a * (*seed) + c) %m;
	return (double) (*seed) / (double) m;
}

__device__ Vector create_vector(double x, double y, double z)
{
	Vector vect;
	vect.x = x;
	vect.y = y;
	vect.z = z;
	return vect;
}

__device__ double dot_product(Vector * in_vect1, Vector * in_vect2)
{
	double out = (in_vect1->x * in_vect2->x);
	out = out + (in_vect1->y * in_vect2->y);
	out = out + (in_vect1->z * in_vect2->z);

	return out;
}

__device__ double norm(Vector * in_vect)
{
	double out = (in_vect->x * in_vect->x) + (in_vect->y * in_vect->y) + (in_vect->z * in_vect->z);
	out = sqrt(out);

	return out;
}

__device__ void scalar_multiply(Vector * in_vect, Vector * out_vect, double scalar)
{
	out_vect->x = in_vect->x * scalar;
	out_vect->y = in_vect->y * scalar;
	out_vect->z = in_vect->z * scalar;
}

__device__ void scalar_divide(Vector * in_vect, Vector * out_vect, double scalar)
{
	out_vect->x = in_vect->x / scalar;
	out_vect->y = in_vect->y / scalar;
	out_vect->z = in_vect->z / scalar;
}

__device__ void vector_add(Vector * in_vect1, Vector * in_vect2, Vector * out_vect)
{
	out_vect->x = in_vect1->x + in_vect2->x;
	out_vect->y = in_vect1->y + in_vect2->y;
	out_vect->z = in_vect1->z + in_vect2->z;
}

__device__ void vector_subtract(Vector * in_vect1, Vector * in_vect2, Vector * out_vect)
{
	out_vect->x = in_vect1->x - in_vect2->x;
	out_vect->y = in_vect1->y - in_vect2->y;
	out_vect->z = in_vect1->z - in_vect2->z;
}

__device__ void direction_sample(Vector * vect, uint64_t * seed)
{

	double rand1 = LCG_random_double(seed);
	double rand2 = LCG_random_double(seed);

	double phi = rand1 * 2 * M_PI;
	double cos_theta = (rand2 * 2) - 1;
	double sin_theta = sqrt(1 - (cos_theta * cos_theta));
	vect->x = sin_theta * cos(phi);
	vect->y = sin_theta * sin(phi);
	vect->z = cos_theta;
}

void write_G(float * G, int num_grid_points)
{
	// writes data to file
	FILE *fp = fopen("plot.bin", "w");

	for (int row_idx=(num_grid_points-1); row_idx>=0; row_idx--)
	{
		for(int col_idx=0; col_idx<num_grid_points; col_idx++)
		 	fprintf(fp, "%f ", G[col_idx + (num_grid_points * row_idx)]);
		fprintf(fp, "\n");
	}

	fclose(fp);

}

__global__ void ray_tracing(float * G, int num_grid_points, int W_max, int W_y, int R, int C_y, int L_x, int L_y, int L_z)
{ 

	// initialize vectors and other variables
	Vector V = create_vector(0, 0, 0);					// light ray direction
	Vector C = create_vector(0, C_y, 0);				// center of sphere
	Vector W = create_vector(0, W_y, 0);				// point where light ray intersects window
	Vector I = create_vector(0, 0, 0);					// point where light ray intersects sphere
	Vector N = create_vector(0, 0, 0);					// unit normal vector at I
	Vector L = create_vector(L_x, L_y, L_z);			// location of light source
	Vector S = create_vector(0, 0, 0);					// direction of light source at I
	Vector temp_vector;									
	double temp;										// represents (V*C)^2 + R^2 - C*C		
	double t;											// t
	double b;											// brightness at I
	int row, col;			

	uint64_t seed = (uint64_t) ((threadIdx.x + blockIdx.x*blockDim.x) * 5356812);

	temp = -1;	// initialize to value less than 0 so loop is entered

	while ((W.x > W_max) || (W.x < (-1*W_max)) || (W.z > W_max) || (W.z < (-1*W_max)) || (temp <= 0))
	{
		direction_sample(&V, &seed);
		scalar_multiply(&V, &W, W_y / V.y);
		temp = dot_product(&V, &C);
		temp = temp * temp;
		temp = temp + (R*R);
		temp = temp - dot_product(&C, &C);
	}

	t = dot_product(&V, &C) - sqrt(temp);
	scalar_multiply(&V, &I, t);

	vector_subtract(&I, &C, &temp_vector);
	scalar_divide(&temp_vector, &N, norm(&temp_vector));

	vector_subtract(&L, &I, &temp_vector);
	scalar_divide(&temp_vector, &S, norm(&temp_vector));

	b = dot_product(&S, &N);
	if (b<0)
		b = 0;

	row = round(num_grid_points * (W.x + W_max) / (2*W_max));
	col = round(num_grid_points * (W.z + W_max) / (2*W_max));

	atomicAdd(&G[col + (num_grid_points * row)], (float) b);

}

int main(int argc, char * argv[])
{

	if (argc != 3)
	{
		printf("2 arguments needed: number of rays, and number of grid points. Exiting.\n");
		return 0;
	}
	int number_of_rays = atoi(argv[1]);
	int num_grid_points = atoi(argv[2]);

	// set timing variables
	double start_time = omp_get_wtime();
	double end_time;

	// set hard-coded parameters
	double W_max = 100;					// width of window
	double W_y = 100;					// window y position
	double R = 60;						// radius of sphere
	double C_y = 200;					// sphere y position 
	double L_x = 500;					// light source x position
	double L_y = -100;					// light source y position
	double L_z = -400;					// light source z position

	// create grid and set all values to 0
	float * G = (float *) malloc(sizeof(float) * num_grid_points * num_grid_points);
	for (int i=0; i<(num_grid_points*num_grid_points); i++)
		G[i] = 0;
	float * dev_G;

	// set device memory
	hipMalloc((void **) &dev_G, sizeof(float) * num_grid_points * num_grid_points);
	printf("hipMalloc status: %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(dev_G, G, sizeof(float) * num_grid_points * num_grid_points, hipMemcpyHostToDevice);
	printf("hipMemcpy status: %s\n", hipGetErrorString(hipGetLastError()));

  	/* launch number_of_rays threads organized in blocks of size THREADS_PER_BLOCK */
  	ray_tracing<<<number_of_rays/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_G, num_grid_points, W_max, W_y, R, C_y, L_x, L_y, L_z);
 	
	printf("kernel: %s\n", hipGetErrorString(hipGetLastError())); 

	hipDeviceSynchronize();

	printf("synchronize status: %s\n", hipGetErrorString(hipGetLastError())); 

  	// copy back memory
  	hipMemcpy(G, dev_G, sizeof(float) * num_grid_points * num_grid_points, hipMemcpyDeviceToHost);
  	printf("hipMemcpy status: %s\n", hipGetErrorString(hipGetLastError())); 

	// print time
	end_time = omp_get_wtime();
	printf("Total Run Time: %f secs\n", end_time - start_time);

	// write G
	printf("Writing G\n");
	write_G(G, num_grid_points);
	free(G);

	return 0;

}

